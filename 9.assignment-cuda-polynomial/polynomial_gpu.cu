
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void polynomial_expansion (float* poly,int degree,int n,float* array) 
{
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  if(idx<n)
    {
    float val=0.0;
      float exp=1.0;
      for(int x=0;x<=degree;++x)
      {
        val+=exp*poly[x];
        exp*=array[idx];
      }
      array[idx]=val;
    }
}

int main(int argc, char* argv[]) 
{
    if(argc<3) 
    {
      std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
      return -1;
    }

  int n=atoi(argv[1]); 
  int degree=atoi(argv[2]);
  int nbiter=1;
    float* array=new float[n];
    float* poly=new float[degree+1];
    for(int i=0;i<n;++i)
  {
      array[i]=1.;
  }

    for(int i=0;i<degree+1;++i)
  {
      poly[i]=1.;
  }

    float *ArrD,*ArrP;

  //start calculating time
    std::chrono::time_point<std::chrono::system_clock> start_time,end_time;
    start_time = std::chrono::system_clock::now();

    hipMalloc(&ArrD,n*sizeof(float));
    hipMalloc(&ArrP,(degree+1)*sizeof(float));

    hipMemcpy(ArrD,array,n*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(ArrP,poly,(degree+1)*sizeof(float),hipMemcpyHostToDevice);

    polynomial_expansion<<<(n+255)/256, 256>>>(ArrP,degree,n,ArrD);
    hipMemcpy(array,ArrD,n*sizeof(float),hipMemcpyDeviceToHost);

    hipFree(ArrD);
    hipFree(ArrP);

    hipDeviceSynchronize();
  {
        bool correct=true;
        int ind;
    for(int i=0;i<n;++i) 
    {
      if(fabs(array[i]-(degree+1))>0.01) 
      {
        correct=false;
        ind=i;
      }
    }
        if(!correct)
        std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  // calculate and print time
    end_time=std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_time=(end_time-start_time)/nbiter;
    std::cout<<n<<" "<<degree<<" "<<elapsed_time.count()<<std::endl;
  
  // free arrays
    delete[] array;
    delete[] poly;

    return 0;
}

