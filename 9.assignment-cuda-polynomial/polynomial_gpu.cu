 
#include <hip/hip_runtime.h>
#include <iostream>
  #include <chrono>

  __global__ void polynomial_expansion (float* poly, int degree,
  int n, float* array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if( idx < n )
    {
    float out = 0.0;
      float xpof = 1.0;
      for ( int x = 0; x < degree; ++x)
      {
        out += xpof * poly[x];
        xpof *= array[idx];
      }
      array[idx] = out;
    }
  }

  int main (int argc, char* argv[]) 
  {
    if (argc < 3) 
    {
      std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
      return -1;
    }

  int n = atoi(argv[1]); 
  int degree = atoi(argv[2]);
  int nbiter = 1;

    float* array = new float[n];
    float* poly = new float[degree+1];
    for (int i=0; i<n; ++i){
      array[i] = 1.;
  }

    for (int i=0; i<degree+1; ++i){
      poly[i] = 1.;
  }

    float *dArr, *dPoly;
    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    hipMalloc(&dArr, n*sizeof(float));
    hipMalloc(&dPoly, (degree+1)*sizeof(float));

    hipMemcpy(dArr, array, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dPoly, poly, (degree+1)*sizeof(float), hipMemcpyHostToDevice);

    polynomial_expansion<<<(n+255)/256, 256>>>(dPoly, degree, n, dArr);
    hipMemcpy(array, dArr, n*sizeof(float), hipMemcpyDeviceToHost);

  
    hipFree(dArr);
    hipFree(dPoly);

    hipDeviceSynchronize();

  {
      bool correct = true;
      int ind;
        for (int i=0; i< n; ++i) {
          if (fabs(array[i]-(degree+1))>0.01) {
            correct = false;
            ind = i;
          }
        }
      if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end-begin)/nbiter;

    std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

    delete[] array;
    delete[] poly;

    return 0;
  }